
#include <hip/hip_runtime.h>
#define DEFAULT_THRESHOLD  8000

#define DEFAULT_FILENAME "BWstop-sign.ppm"
#define BLOCK_SIZE 32
#define TILE_SIZE 32
__global__ void sobel( int xd_size, int yd_size, int maxdval, int d_thresh, unsigned int *input , int * output)
{

	int sum1,sum2,magnitude;
	
int i = blockIdx.y * blockDim.y + threadIdx.y; 
	int j = blockIdx.x * blockDim.x + threadIdx.x; 
	int tid_x = threadIdx.y; 
	int tid_y = threadIdx.x;
__shared__ int inter[(TILE_SIZE) * (TILE_SIZE)];
	
	
	if ((i < yd_size) && (j < xd_size))
        {
        output[i * xd_size + j] = 0;
        }
        __syncthreads();
    inter[tid_x * TILE_SIZE + tid_y]  = 0; 
	__syncthreads();
	inter[tid_x * TILE_SIZE + tid_y] = input[i * (xd_size) + j];
	__syncthreads();
	if (i > 0 && j > 0 && i < yd_size - 1 && j < xd_size - 1){	
	
		if ((tid_x > 0) && (tid_x < TILE_SIZE - 1)  && (tid_y > 0) && (tid_y < TILE_SIZE - 1))
        	{
        	int offset = i * xd_size + j;
        	int shared_a = inter[ TILE_SIZE * (tid_x-1) + tid_y+1];
        	int shared_b = inter[ TILE_SIZE * (tid_x-1) + tid_y-1 ];
        	int shared_c = inter[ TILE_SIZE * (tid_x+1) + tid_y+1];

       		 sum1 =  shared_a - shared_b+shared_c + 2 * inter[ TILE_SIZE * (tid_x)   + tid_y+1 ] - 2 * inter[ TILE_SIZE*(tid_x)   + tid_y-1 ] - inter[ TILE_SIZE*(tid_x+1) + tid_y-1 ];

        	sum2 = shared_a+shared_b-shared_c + 2 * inter[ TILE_SIZE * (tid_x-1) + tid_y ]  - inter[TILE_SIZE * (tid_x+1) + tid_y-1 ] - 2 * inter[ TILE_SIZE * (tid_x+1) + tid_y ];
		magnitude=sum1*sum1+sum2*sum2;
	
		int e_ig =0;
		if(magnitude>d_thresh){
          	e_ig=255;
		}
       output[offset]=e_ig;}
       __syncthreads();
		if ((i == blockIdx.y * blockDim.y + blockDim.y - 1) || (j == blockIdx.x * blockDim.x + blockDim.x - 1) ||  (i == blockIdx.y * blockDim.y) || (j == blockIdx.x * blockDim.x))
		{
		int offset = i * xd_size + j;
		int golbal_a = input[ xd_size * (i-1) + j+1];
		int golbal_b = input[ xd_size * (i-1) + j-1 ];
		int golbal_c = input[ xd_size *	(i+1) + j+1];

       		 sum1 =  golbal_a - golbal_b+golbal_c+ 2 * input[ xd_size * (i)   + j+1 ] - 2 * input[ xd_size*(i)   + j-1 ] -input[ xd_size*(i+1) + j-1 ];

       		 sum2 = golbal_a+golbal_b + 2 * input[ xd_size * (i-1) + j ] - input[xd_size * (i+1) + j-1 ] - 2 * input[ xd_size * (i+1) + j ] - golbal_c;

		magnitude=sum1*sum1+sum2*sum2;
		int e_ig=0;
		if(magnitude>d_thresh){
        		e_ig=255;
        	}
        output[offset]=e_ig;
    }
      			  __syncthreads();
}               
}

